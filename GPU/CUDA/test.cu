#include "hip/hip_runtime.h"
__global__ void setHandle(cudaGraphConditionalHandle handle)
{
    cudaGraphSetConditional(handle, value);
}

void graphSetup() {
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t node;
    void *kernelArgs[1];
    int value = 1;

    hipGraphCreate(&graph, 0);

    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);

    // Use a kernel upstream of the conditional to set the handle value
    hipGraphNodeParams params = { hipGraphNodeTypeKernel };
    params.kernel.func = (void *)setHandle;
    params.kernel.gridDim.x = params.kernel.gridDim.y = params.kernel.gridDim.z = 1;
    params.kernel.blockDim.x = params.kernel.blockDim.y = params.kernel.blockDim.z = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &handle;
    hipGraphAddNode(&node, graph, NULL, 0, &params);

    hipGraphNodeParams cParams = { hipGraphNodeTypeConditional };
    cParams.conditional.handle = handle;
    cParams.conditional.type   = cudaGraphCondTypeIf;
    cParams.conditional.size   = 1;
    hipGraphAddNode(&node, graph, &node, 1, &cParams);

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    // Populate the body of the conditional node
    hipGraphAddNode(&node, bodyGraph, NULL, 0, &params);

    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    hipGraphLaunch(graphExec, 0);
    hipDeviceSynchronize();

    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
}