// #include <stdio.h>

#include <hip/hip_runtime.h>
#include <iostream>


#define BLOCK_SIZE 32

typedef struct {
    int width;
    int stride;
    int height;
    float* elements;
}Matrix;

__device__ void SetElement(Matrix &mat, int row, int col, float v)
{
    if(row >= mat.height) {
        return ;
    }
    if(col >= mat.width) {
        return ;
    }
    mat.elements[row * mat.stride + col] = v;
}

__device__ float GetElement(Matrix &mat, int row, int col)
{
    if(row >= mat.height) {
        return 0;
    }
    if(col >= mat.width) {
        return 0;
    }
    return mat.elements[row * mat.stride + col];
}

__device__ Matrix GetSubMatrix(Matrix &mat, int idx, int idy)
{
    Matrix sub;

    if (idx*BLOCK_SIZE > mat.width) {
        sub.width = 0;
        return sub;
    }

    if (idy*BLOCK_SIZE > mat.height) {
        sub.height = 0;
        return sub;
    }

    sub.width = min(BLOCK_SIZE, mat.width - idx * BLOCK_SIZE);
    sub.height = min(BLOCK_SIZE, mat.height - idy * BLOCK_SIZE);
    // sub.width = BLOCK_SIZE;
    // sub.height = BLOCK_SIZE;
    sub.stride = mat.stride;
    sub.elements = &mat.elements[idy * BLOCK_SIZE * mat.stride
                            + idx * BLOCK_SIZE];
    return sub;
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    int idn = blockIdx.x;
    int idm = blockIdx.y;

    Matrix subC = GetSubMatrix(C, idn, idm);

    float sum = 0;
    int kloop = (A.width + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for(int idk = 0; idk < kloop; idk++) {
        __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

        Matrix subA = GetSubMatrix(A, idk, idm);
        Matrix subB = GetSubMatrix(B, idn, idk);

        sharedA[threadIdx.y][threadIdx.x] = GetElement(subA, threadIdx.y, threadIdx.x);
        sharedB[threadIdx.y][threadIdx.x] = GetElement(subB, threadIdx.y, threadIdx.x);
        // SetElement(subA, threadIdx.y, threadIdx.x, )
        __syncthreads();
        for(int k = 0; k < BLOCK_SIZE; k++) {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }
        __syncthreads();

    }
    SetElement(subC, threadIdx.y, threadIdx.x, sum);
}

void MatMul(const Matrix &A, const Matrix &B, Matrix &C)
{
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    d_A.stride = A.stride;
    int sizeA = d_A.stride * d_A.height * sizeof(float);
    hipMalloc(&d_A.elements, sizeA);
    hipMemcpy(d_A.elements, A.elements, sizeA, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    d_B.stride = B.stride;
    int sizeB = d_B.stride * d_B.height * sizeof(float);
    hipMalloc(&d_B.elements, sizeB);
    hipMemcpy(d_B.elements, B.elements, sizeB, hipMemcpyHostToDevice);

    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    d_C.stride = C.stride;
    int sizeC = d_C.stride * d_C.height * sizeof(float);
    hipMalloc(&d_C.elements, sizeC);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((B.width + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (A.height + BLOCK_SIZE - 1) / BLOCK_SIZE
    );
    MatMulKernel<<<grid, block>>>(d_A, d_B, d_C);

    hipMemcpy(C.elements, d_C.elements, sizeC, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

int main()
{
    int M = BLOCK_SIZE * 2 + 1;
    int K = BLOCK_SIZE * 3 + 1;
    int N = BLOCK_SIZE * 4 + 1;

    Matrix A;
    A.width = K;
    A.stride = K;
    A.height = M;
    A.elements = (float*)malloc(M * K * sizeof(float));

    Matrix B;
    B.width = N;
    B.stride = N;
    B.height = K;
    B.elements = (float*)malloc(K * N * sizeof(float));

    Matrix C;
    C.width = N;
    C.stride = N;
    C.height = M;
    C.elements = (float*)malloc(M * N * sizeof(float));

    Matrix golden;
    golden.width = N;
    golden.stride = N;
    golden.height = M;
    golden.elements = (float*)malloc(M * N * sizeof(float));

    for(int i = 0; i < M * K; i ++) {
        A.elements[i] = 1;
    }
    for(int i = 0; i < N * K; i ++) {
        B.elements[i] = 1;
    }

    MatMul(A, B, C);

    for(int m = 0; m < M; m++) {
        for(int n = 0; n < N; n++) {
            for(int k = 0; k < K; k++) {
                golden.elements[m * C.stride + n] +=
                    A.elements[m * A.stride + k] *
                        B.elements[k * B.stride + n];
            }
        }
    }

    for(int m = 0; m < M; m++) {
        for(int n = 0; n < N; n++) {
            if(golden.elements[m * C.stride + n] != C.elements[m * C.stride + n]) {
                std::cout << "[" << m << ", " << n << "]: "
                        << "golden: " << golden.elements[m * C.stride + n] << ", "
                        << "c: " << C.elements[m * C.stride + n]
                        << std::endl;
            }
        }
    }
    std::cout << "end" << std::endl;

    return 0;
}